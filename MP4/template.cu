#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
const int MASK_RADIUS = 1;
const int MASK_SIZE = 3;
const int TILED_SIZE = 3;
const int CACHE_SIZE = TILED_SIZE + MASK_SIZE - 1;
//@@ Define constant memory for device kernel here
__constant__ float MASK[27];
__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  const int bx = blockIdx.x;
  const int by = blockIdx.y;
  const int bz = blockIdx.z;
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int tz = threadIdx.z;
  __shared__ float matrix[CACHE_SIZE][CACHE_SIZE][CACHE_SIZE];

  int output_x = bx * TILED_SIZE + tx;
  int output_y = by * TILED_SIZE + ty;
  int output_z = bz * TILED_SIZE + tz;

  int input_x = output_x - MASK_RADIUS;
  int input_y = output_y - MASK_RADIUS;
  int input_z = output_z - MASK_RADIUS;

  if (input_x >= 0 && input_x < x_size && input_y >= 0 && input_y < y_size &&
      input_z >= 0 && input_z < z_size) {
    matrix[tz][ty][tx] =
        input[input_z * y_size * x_size + input_y * x_size + input_x];
  } else {
    matrix[tz][ty][tx] = 0.0;
  }

  __syncthreads();

  float value = 0.0;
  if (tx < TILED_SIZE && ty < TILED_SIZE && tz < TILED_SIZE) {
    for (int i = 0; i < MASK_SIZE; i++) {
      for (int j = 0; j < MASK_SIZE; j++) {
        for (int k = 0; k < MASK_SIZE; k++) {
          value +=
              MASK[i * TILED_SIZE * TILED_SIZE + j * TILED_SIZE + k] *
              matrix[tz + i][ty + j][tx + k];
        }
      }
    }
    if (output_x < x_size && output_y < y_size && output_z < z_size)
      output[output_z * x_size * y_size + output_y * x_size + output_x] = value;
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel = (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
 
  
  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  hipMalloc((void**) &deviceInput, sizeof(float) * (inputLength-3));
  hipMalloc((void**) &deviceOutput, sizeof(float) * (inputLength-3));
  wbTime_stop(GPU, "Doing GPU memory allocation");
  
  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do not need to be copied to the gpu
  hipMemcpy(deviceInput, hostInput+3, sizeof(float) * z_size * y_size * x_size, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(MASK), hostKernel, sizeof(float) * kernelLength, 0, hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 DimGrid(ceil((1.0 * x_size)/TILED_SIZE), ceil((1.0 * y_size)/TILED_SIZE), ceil((1.0 * z_size)/TILED_SIZE));
  dim3 DimBlock(CACHE_SIZE, CACHE_SIZE, CACHE_SIZE);
  conv3d <<< DimGrid, DimBlock >>> (deviceInput, deviceOutput, z_size, y_size, x_size);
  //@@ Launch the GPU kernel here
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy((hostOutput+3), deviceOutput, sizeof(float) * z_size * y_size * x_size, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}